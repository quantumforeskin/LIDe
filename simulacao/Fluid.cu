#include "hip/hip_runtime.h"
#include"Fluid.h"

__global__ void cudacop(float *c1, float *c2, float *c3, float *c4, float *c5){

  int index=threadIdx.x;

  c1[index]=0;
  c2[index]=0;
  c3[index]=0;
  c4[index]=0;
  c5[index]=0;

}

void Fluid::Evolution1c(const char *output, ld time, int steps){

  float *caux1, *caux2, *caux3, *caux4, *caux5;
  float  *aux1,  *aux2,  *aux3,  *aux4,  *aux5;

  aux1=new float[numberx];aux2=new float[numberx];
  aux3=new float[numberx];aux4=new float[numberx];
  aux5=new float[numberx];

  if(hipSuccess!=hipMalloc((void**)&caux1,numberx*sizeof(float)))cout<<"oi"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux2,numberx*sizeof(float)))cout<<"oi"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux3,numberx*sizeof(float)))cout<<"oi"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux4,numberx*sizeof(float)))cout<<"oi"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux5,numberx*sizeof(float)))cout<<"oi"<<endl;

  cudacop<<<1,numberx>>>(caux1,caux2,caux3,caux4,caux5);

  if(hipSuccess!=hipMemcpy(aux1,caux1,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"oi"<<endl;
  if(hipSuccess!=hipMemcpy(aux2,caux2,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"oi"<<endl;
  if(hipSuccess!=hipMemcpy(aux3,caux3,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"oi"<<endl;
  if(hipSuccess!=hipMemcpy(aux4,caux4,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"oi"<<endl;
  if(hipSuccess!=hipMemcpy(aux5,caux5,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"oi"<<endl;

  cout<<aux1[0]<<" "<<aux1[1]<<endl;
  cout<<aux2[0]<<" "<<aux2[1]<<endl;
  cout<<aux3[0]<<" "<<aux3[1]<<endl;
  cout<<aux4[0]<<" "<<aux4[1]<<endl;
  cout<<aux5[0]<<" "<<aux5[1]<<endl;

}
