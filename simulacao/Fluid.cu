#include "hip/hip_runtime.h"
#include"Fluid.h"

//___________________________________________________________
__global__ void callcuda(float *cad, float *ct, float *cx, int *cnumber, float *caux1, float *caux2, float *caux3, float *caux4, float *caux5, float *crho, float *crhou, float *crhov, float *crhow, float *cenergy){

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int undindex=index-1, topindex=index+1;
  if(topindex>cnumber[0]-1)topindex=cnumber[0]-1;
  if(undindex<0)undindex=0;

  float uu=crhou[undindex]/crho[undindex];
  float ut=crhou[topindex]/crho[topindex];
  float undpressure=(cad[0]-1)*(cenergy[undindex]-crho[undindex]*uu*uu/2);
  float toppressure=(cad[0]-1)*(cenergy[topindex]-crho[topindex]*ut*ut/2);

  /*  calculates f1_r and f1_l  */
  float f10=crhou[undindex];
  float f11=crhou[undindex]*uu+undpressure;
  float f12=crhov[undindex]*uu;
  float f13=crhow[undindex]*uu;
  float f14=(cenergy[undindex]+undpressure)*uu;

  float f20=crhou[topindex];
  float f21=crhou[topindex]*ut+toppressure;
  float f22=crhov[topindex]*ut;
  float f23=crhow[topindex]*ut;
  float f24=(cenergy[topindex]+toppressure)*ut;

  /*  calculates evolution  */
  caux1[index]=crho[index]   -ct[0]/(cx[0])*(f10-f20);
  caux2[index]=crhou[index]  -ct[0]/(cx[0])*(f11-f21);
  caux3[index]=crhov[index]  -ct[0]/(cx[0])*(f12-f22);
  caux4[index]=crhow[index]  -ct[0]/(cx[0])*(f13-f23);
  caux5[index]=cenergy[index]-ct[0]/(cx[0])*(f14-f24);

}

//___________________________________________________________
void Fluid::Evolution1c(const char *output, long double time, int nsteps){

 //  runs evolution of fluid through time with nsteps steps in time, 
 // writing the result into output, using simple numeric method

  ofstream aaa(output);
  float stept=time/(nsteps-1), stepx=(upperx-bottomx)/(numberx-1);

  for(int i=0;i<numberx;++i)if(i%20==0)// writes initial condition
    aaa<<"0\t"<<i*stepx<<"\t"<<rho[i]<<"\t"<<rhou[i]<<"\t"<<rhov[i]<<"\t"<<rhow[i]<<"\t"<<Energy[i]<<"\t"<<Pressure(i)<<endl;

  /*  Allocates memory in CUDA and copies static variables  */

  int *cnumber;
  float *cad, *ct, *cx;
  float *crho, *crhou,*crhov,*crhow,*cenergy;
  float *caux1,*caux2,*caux3,*caux4,*caux5;

  if(hipSuccess!=hipMalloc((void**)&cnumber,sizeof(int)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMemcpy(cnumber,&numberx,sizeof(int),hipMemcpyHostToDevice))cout<<"copia"<<endl;
  if(hipSuccess!=hipMalloc((void**)&cad,       sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMemcpy(cnumber,&addiabatic,sizeof(float),hipMemcpyHostToDevice))cout<<"copia"<<endl;
  if(hipSuccess!=hipMalloc((void**)&ct,sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMemcpy(ct,&stept,  sizeof(float),hipMemcpyHostToDevice))cout<<"copia"<<endl;
  if(hipSuccess!=hipMalloc((void**)&cx,sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMemcpy(cx,&stepx,  sizeof(float),hipMemcpyHostToDevice))cout<<"copia"<<endl;

  if(hipSuccess!=hipMalloc((void**)&caux1  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux2  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux3  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux4  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&caux5  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&crho   ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&crhou  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&crhov  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&crhow  ,numberx*sizeof(float)))cout<<"memoria"<<endl;
  if(hipSuccess!=hipMalloc((void**)&cenergy,numberx*sizeof(float)))cout<<"memoria"<<endl;

  /*  Runs evolution  */
  for(int i=1;i<nsteps;++i){

    /*  copies current status to cuda  */
    if(hipSuccess!=hipMemcpy(crho   ,rho   ,numberx*sizeof(float),hipMemcpyHostToDevice))cout<<"copytodevice"<<endl;
    if(hipSuccess!=hipMemcpy(crhou  ,rhou  ,numberx*sizeof(float),hipMemcpyHostToDevice))cout<<"copytodevice"<<endl;
    if(hipSuccess!=hipMemcpy(crhov  ,rhov  ,numberx*sizeof(float),hipMemcpyHostToDevice))cout<<"copytodevice"<<endl;
    if(hipSuccess!=hipMemcpy(crhow  ,rhow  ,numberx*sizeof(float),hipMemcpyHostToDevice))cout<<"copytodevice"<<endl;
    if(hipSuccess!=hipMemcpy(cenergy,Energy,numberx*sizeof(float),hipMemcpyHostToDevice))cout<<"copytodevice"<<endl;

    //int threadsize=64;// calculates evolution
    callcuda<<<1,numberx>>>(cad,ct,cx,cnumber,caux1,caux2,caux3,caux4,caux5,crho,crhou,crhov,crhow,cenergy);

    /*  copies result into host  */
    if(hipSuccess!=hipMemcpy(rho,&caux1,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"copyfromdevice"<<endl;
    if(hipSuccess!=hipMemcpy(rhou,&caux2,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"copyfromdevice"<<endl;
    if(hipSuccess!=hipMemcpy(rhov,&caux3,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"copyfromdevice"<<endl;
    if(hipSuccess!=hipMemcpy(rhow,&caux4,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"copyfromdevice"<<endl;
    if(hipSuccess!=hipMemcpy(Energy,&caux5,numberx*sizeof(float),hipMemcpyDeviceToHost))cout<<"copyfromdevice"<<endl;

    for(int j=0;j<numberx;++j)if(j%20==0)// writes output
      aaa<<i*stept<<"\t"<<j*stepx<<"\t"<<rho[j]<<"\t"<<rhou[j]<<"\t"<<rhov[j]<<"\t"<<rhow[j]<<"\t"<<Energy[j]<<"\t"<<Pressure(j)<<endl;

  }

}

