#include "hip/hip_runtime.h"
#include"Fluid.h"

//___________________________________________________________
__global__ void callcuda(float *cad, float *ct, float *cx, int *cnumber, float *caux1, float *caux2, float *caux3, float *caux4, float *caux5, float *crho, float *crhou, float *crhov, float *crhow, float *cenergy){

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int undindex=index-1, topindex=index+1;
  if(topindex>*cnumber-1)topindex=*cnumber-1;
  if(undindex<0)undindex=0;

  float uu=crhou[undindex]/crho[undindex];
  float ut=crhou[topindex]/crho[topindex];
  float undpressure=(*cad-1)*(cenergy[undindex]-crho[undindex]*uu*uu/2);
  float toppressure=(*cad-1)*(cenergy[topindex]-crho[topindex]*ut*ut/2);

  float f10=crhou[undindex];
  float f11=crhou[undindex]*uu+undpressure;
  float f12=crhov[undindex]*uu;
  float f13=crhow[undindex]*uu;
  float f14=(cenergy[undindex]+undpressure)*uu;
  float f20=crhou[topindex];
  float f21=crhou[topindex]*ut+toppressure;
  float f22=crhov[topindex]*ut;
  float f23=crhow[topindex]*ut;
  float f24=(cenergy[topindex]+toppressure)*ut;

  caux1[index]=crho[index]   -*ct/(*cx)*(f10-f20);
  caux2[index]=crhou[index]  -*ct/(*cx)*(f11-f21);
  caux3[index]=crhov[index]  -*ct/(*cx)*(f12-f22);
  caux4[index]=crhow[index]  -*ct/(*cx)*(f13-f23);
  caux5[index]=cenergy[index]-*ct/(*cx)*(f14-f24);

}

//___________________________________________________________
void Fluid::Evolution1c(const char *output, double time, int nsteps){

 //  runs evolution of fluid through time with nsteps steps in time, 
 // writing the result into output, using simple numeric method

  ofstream aaa(output);
  float stept=time/(nsteps-1), stepx=(upperx-bottomx)/(numberx-1);

  for(int i=0;i<numberx;++i)
    aaa<<"0\t"<<i*stepx<<"\t"<<rho[i]<<"\t"<<rhou[i]<<"\t"<<rhov[i]<<"\t"<<rhow[i]<<"\t"<<Energy[i]<<"\t"<<Pressure(i)<<endl;

  int *cnumber;
  float *cad, *ct, *cx;
  float *crho, *crhou,*crhov,*crhow,*cenergy;
  float *caux1,*caux2,*caux3,*caux4,*caux5;

  hipMalloc((void**)&cnumber,sizeof(int));
  hipMemcpy(cnumber,&numberx,sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&cad,       sizeof(float));
  hipMemcpy(cnumber,&addiabatic,sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&ct,sizeof(float));
  hipMemcpy(ct,&stept,  sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&cx,sizeof(float));
  hipMemcpy(cx,&stepx,  sizeof(float),hipMemcpyHostToDevice);

  hipMalloc((void**)&caux1  ,numberx*sizeof(float));
  hipMalloc((void**)&caux2  ,numberx*sizeof(float));
  hipMalloc((void**)&caux3  ,numberx*sizeof(float));
  hipMalloc((void**)&caux4  ,numberx*sizeof(float));
  hipMalloc((void**)&caux5  ,numberx*sizeof(float));
  hipMalloc((void**)&crho   ,numberx*sizeof(float));
  hipMalloc((void**)&crhou  ,numberx*sizeof(float));
  hipMalloc((void**)&crhov  ,numberx*sizeof(float));
  hipMalloc((void**)&crhow  ,numberx*sizeof(float));
  hipMalloc((void**)&cenergy,numberx*sizeof(float));

  for(int i=1;i<nsteps;++i){

    hipMemcpy(crho   ,&rho   ,numberx*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(crhou  ,&rhou  ,numberx*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(crhov  ,&rhov  ,numberx*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(crhow  ,&rhow  ,numberx*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(cenergy,&Energy,numberx*sizeof(float),hipMemcpyHostToDevice);

    int threadsize=64;
    callcuda<<<numberx/threadsize+1,threadsize>>>(cad,ct,cx,cnumber,caux1,caux2,caux3,caux4,caux5,crho,crhou,crhov,crhow,cenergy);

    hipMemcpy(rho,&caux1,numberx*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(rhou,&caux2,numberx*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(rhov,&caux3,numberx*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(rhow,&caux4,numberx*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(Energy,&caux5,numberx*sizeof(float),hipMemcpyDeviceToHost);

    for(int j=0;j<numberx;++j)
      aaa<<i*stept<<"\t"<<j*stepx<<"\t"<<rho[j]<<"\t"<<rhou[j]<<"\t"<<rhov[j]<<"\t"<<rhow[j]<<"\t"<<Energy[j]<<"\t"<<Pressure(j)<<endl;

  }

}

